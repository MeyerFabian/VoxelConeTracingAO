#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <src/Utilities/errorUtils.h>
#include "fillOctree.cuh"


const int maxNodePoolSize = 1024;
int volumeResolution = 384;

bool constantMemoryValid = false;   // the flag indicates wheather a kernel is allowed to use the constantNodePool
__constant__ node constNodePool[maxNodePoolSize];
__constant__ int constVolumeResolution[1];
__device__ unsigned int globalNodePoolCounter = 0;
__device__ unsigned int globalBrickPoolCounter = 0;

surface<void, cudaSurfaceType3D> surfRef;

hipError_t setVolumeResulution(int resolution)
{
    volumeResolution = resolution;
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constVolumeResolution), &resolution, sizeof(int));
    return errorCode;
}

__device__
unsigned int getBit(unsigned int value, unsigned int position)
{
    return (value >> (position-1)) & 1u;
}

__device__
void setBit(unsigned int &value, unsigned int position)
{
    value |= (1u << (position-1));
}

__device__
void unSetBit(unsigned int &value, unsigned int position)
{
    value &= ~(1u << (position-1));
}

__global__
void testFilling(dim3 texture_dim)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if(x >= texture_dim.x || y >= texture_dim.y || z >= texture_dim.z)
    {
        return;
    }

    uchar4 element = make_uchar4(255, 255, 255, 255);
    surf3Dwrite(element, surfRef, x*sizeof(uchar4), y, z);
}

__global__
void clearNodePoolKernel(node *nodePool, int poolSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= poolSize)
        return;

    nodePool[i].nodeTilePointer = 0;
    nodePool[i].value = 0;
}

__global__
void clearCounter()
{
    globalNodePoolCounter = 0;
    globalBrickPoolCounter = 0;
}

__device__ uint3 getBrickCoords(unsigned int brickAdress, unsigned int brickPoolSideLength, unsigned int brickSideLength = 3)
{
    uint3 coords;
    coords.x = brickAdress / (brickPoolSideLength*brickPoolSideLength);
    coords.y = (brickAdress / brickPoolSideLength) % brickPoolSideLength;
    coords.z = brickAdress % brickPoolSideLength;

    //TODO: consider brickSideLength for variable brick size
    coords.x = coords.x*2+1;
    coords.y = coords.y*2+1;
    coords.z = coords.z*2+1;

    return coords;
}

__device__ unsigned int encodeBrickCoords(uint3 coords)
{
    return (0x000003FF & coords.x) << 20U | (0x000003FF & coords.y) << 10U | (0x000003FF & coords.z);
}

__device__ void fillBrick(uint3 brickCoords, float3 voxelPosition)
{
    // TODO: calculate the responding voxel within the brickpool. update the shared atomic counter for duplicate voxels
}

__global__ void insertVoxelsInLastLevel(node *nodePool, uint1 *positionBuffer, uchar4* colorBufferDevPointer, unsigned int maxLevel, int fragmentListSize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= fragmentListSize)
        return;

    const unsigned int mask_bits = 0x000003FF;
    unsigned int codedPosition = positionBuffer[index].x;

    float3 position;
    // dont forget the .f for casting reasons :P
    position.x = ((codedPosition) & (mask_bits)) / 1024.f;
    position.y = ((codedPosition >> 10) & (mask_bits)) / 1024.f;
    position.z = ((codedPosition >> 20) & (mask_bits)) / 1024.f;


    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;
    unsigned int offset=0;
    unsigned int nodeTile = 0;
    unsigned int value = 0;
    if(index == 0)
        printf("follow fragment 1 during octree processing: \n");

    for (int i = 0; i < maxLevel; i++)
    {
        uint3 nextOctant = make_uint3(0, 0, 0);
        // determine octant for the given voxel
        if(i != 0)
        {
            nextOctant.x = static_cast<unsigned int>(2 * position.x);
            nextOctant.y = static_cast<unsigned int>(2 * position.y);
            nextOctant.z = static_cast<unsigned int>(2 * position.z);
        }

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2 * nextOctant.y + 4 * nextOctant.z;
        offset = nodeOffset + childPointer * 8;

        nodeTile = nodePool[offset].nodeTilePointer;

        childPointer = nodeTile & 0x3fffffff;

        if(index == 0)
        {
            printf("level: %d\n", i);
            printf("childPointer: %d\n", childPointer);
            printf("maxDivide bit: %d\n", getBit(nodeTile, 32));
        }

        if(i != 0)
        {
            position.x = 2 * position.x - nextOctant.x;
            position.y = 2 * position.y - nextOctant.y;
            position.z = 2 * position.z - nextOctant.z;
        }
    }
    if(index == 0)
    {
        printf("######################################## \n");
    }

     value = nodePool[offset].value;
}

__global__ void markNodeForSubdivision(node *nodePool, int poolSize, int maxLevel, uint1* positionBuffer, int fragmentListSize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= fragmentListSize)
        return;

    // mask to get 10 bit position coords
    const unsigned int mask_bits = 0x000003FF;
    unsigned int codedPosition = positionBuffer[index].x;

    float3 position;
    // dont forget the .f for casting reasons :P
    position.x = ((codedPosition) & (mask_bits)) / 1024.f;
    position.y = ((codedPosition >> 10) & (mask_bits)) / 1024.f;
    position.z = ((codedPosition >> 20) & (mask_bits)) / 1024.f;


    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    for(int i=0;i<=maxLevel;i++)
    {
        uint3 nextOctant = make_uint3(0, 0, 0);
        // determine octant for the given voxel
        unsigned int offset = 0;
        if(i != 0)
        {
            nextOctant.x = static_cast<unsigned int>(2 * position.x);
            nextOctant.y = static_cast<unsigned int>(2 * position.y);
            nextOctant.z = static_cast<unsigned int>(2 * position.z);

            // make the octant position 1D for the linear memory
            nodeOffset = nextOctant.x + 2 * nextOctant.y + 4 * nextOctant.z;
            offset = nodeOffset + childPointer * 8;
        }

        // the maxdivide bit indicates wheather the node has children 1 means has children 0 means does not have children
        unsigned int nodeTile = nodePool[offset].nodeTilePointer;
        __syncthreads();
        unsigned int maxDivide = getBit(nodeTile,32);

        if(maxDivide == 0)
        {
            // as the node has no children we set the second bit to 1 which indicates that memory should be allocated
            setBit(nodeTile,31); // possible race condition but it is not importatnt in our case
            nodePool[offset].nodeTilePointer = nodeTile;
            __syncthreads();
            break;
        }
        else
        {
            // if the node has children we read the pointer to the next nodetile
            childPointer = nodeTile & 0x3fffffff;
        }

        if(i!=0)
        {
            position.x = 2 * position.x - nextOctant.x;
            position.y = 2 * position.y - nextOctant.y;
            position.z = 2 * position.z - nextOctant.z;
        }
    }
}

__global__ void reserveMemoryForNodes(node *nodePool, int maxNodes, int level, unsigned int* counter, unsigned int brickPoolResolution, unsigned int brickResolution, int lastLevel)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= maxNodes)
        return;

    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    uint3 octants[8];
    octants[0] = make_uint3(0,0,0);
    octants[1] = make_uint3(0,0,1);
    octants[2] = make_uint3(0,1,0);
    octants[3] = make_uint3(0,1,1);
    octants[4] = make_uint3(1,0,0);
    octants[5] = make_uint3(1,0,1);
    octants[6] = make_uint3(1,1,0);
    octants[7] = make_uint3(1,1,1);

    uint3 nextOctant;
    unsigned int octantIdx = 0;

    for (int i = 0; i <=level; i++)
    {
        if(i==0)
            octantIdx = 0;
        else
            octantIdx = (index / static_cast<unsigned int>(pow(8.f, static_cast<float>(i-1)))) % 8;

        nextOctant = octants[octantIdx];

       // if(level == 3)
          //  printf("octant %d => index: %d i: %d\n", octantIdx, index, i);

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2 * nextOctant.y + 4 * nextOctant.z;

        unsigned int offset = nodeOffset + childPointer * 8;

        unsigned int pointer = nodePool[offset].nodeTilePointer;
        unsigned int value = nodePool[offset].value;
        __syncthreads();    //make sure all threads have a valid nodeTilePointer

        unsigned int reserve = getBit(pointer, 31);
        unsigned int maxDivided = getBit(pointer, 32);
        if (reserve == 1)
        {
            //if(level==3)
              //  printf("reserve\n");
            // increment the global nodecount and allocate the memory in our
            unsigned int adress = atomicAdd(counter, 1) + 1;
            unsigned int brickAdress = atomicAdd(&globalBrickPoolCounter, 1);

            pointer = (adress & 0x3fffffff) | pointer;
            value = encodeBrickCoords(getBrickCoords(brickAdress, brickPoolResolution, brickResolution));

            // set the first bit to 1. this indicates, that we use the texture brick instead of a constant value as color.
            setBit(value, 32);
            setBit(pointer, 32);

            // make sure we don't reserve the same nodeTile next time :)
            unSetBit(pointer, 31);

            if(lastLevel == 1)
                unSetBit(pointer,32);

            nodePool[offset].nodeTilePointer = pointer;
            nodePool[offset].value = value;

            __syncthreads();
            break;
        }
        else
        {
            // traverse further
            childPointer = pointer & 0x3fffffff;
           // if(level==3)
            //    printf("getChild %d\n", childPointer);
        }
    }

}

hipError_t updateBrickPool(hipArray_t &brickPool, dim3 textureDim)
{
    hipError_t errorCode;

    hipChannelFormatDesc channelDesc;
    errorCode = hipGetChannelDesc(&channelDesc, brickPool);

    if(errorCode != hipSuccess)
        return errorCode;

    errorCode = cudaBindSurfaceToArray(&surfRef, brickPool, &channelDesc);

    if(errorCode != hipSuccess)
        return errorCode;

    dim3 block_dim(4, 4, 4);
    dim3 grid_dim(textureDim.x/block_dim.x, textureDim.y/block_dim.y, textureDim.z/block_dim.z);
    testFilling<<<grid_dim, block_dim>>>(textureDim);

    return hipSuccess;
}

hipError_t updateNodePool(uchar4* colorBufferDevPointer, node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 64;
    int blockCount = poolSize / threadsPerBlock;


    struct node *node_h = (struct node*)malloc(sizeof(node) * poolSize);

    errorCode = hipMemcpy(node_h, nodePool, sizeof(node) * poolSize, hipMemcpyDeviceToHost);

    if(errorCode != hipSuccess)
        return errorCode;


    free(node_h);

    return hipSuccess;
}

hipError_t copyNodePoolToConstantMemory(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constNodePool),nodePool,sizeof(node)*poolSize,0,hipMemcpyDeviceToDevice);

    if(errorCode != hipSuccess)
    {
        constantMemoryValid = false;
        return errorCode;
    }
    else
    {
        constantMemoryValid = true;
        return errorCode;
    }
}

hipError_t buildSVO(node *nodePool,
                     int poolSize,
                     hipArray_t *brickPool,
                     dim3 textureDim,
                     uint1* positionDevPointer,
                     uchar4* colorBufferDevPointer,
                     uchar4* normalDevPointer,
                     int fragmentListSize)
{
    hipError_t errorCode = hipSuccess;
    // calculate maxlevel
    int maxLevel = static_cast<int>(log((volumeResolution*volumeResolution*volumeResolution)/27)/log(8));

    printf("max level: %d \n", maxLevel);

    dim3 block_dim(32, 0, 0);
    dim3 grid_dim(fragmentListSize/block_dim.x, 0, 0);

    int threadsPerBlock = 64;
    int blockCount = fragmentListSize / threadsPerBlock;


    unsigned int *h_counter = new unsigned int[1];
    unsigned int *d_counter;
    *h_counter = 0;

    hipMalloc(&d_counter, sizeof(int));
    hipMemcpy(d_counter,h_counter,sizeof(unsigned int),hipMemcpyHostToDevice);

    clearCounter<<<1,1>>>();
    hipDeviceSynchronize();

    printf("counter: %d\n", *h_counter);
    int lastLevel = 0;

    for(int i=0;i<maxLevel;i++)
    {
        markNodeForSubdivision<<<blockCount, threadsPerBlock>>>(nodePool, poolSize, i, positionDevPointer, fragmentListSize);
        hipDeviceSynchronize();
        unsigned int maxNodes = static_cast<unsigned int>(pow(8,i));

        const int threadPerBlockReserve = 32;
        int blockCountReserve = maxNodes;

        if(maxNodes >= threadPerBlockReserve)
            blockCountReserve = maxNodes / threadPerBlockReserve;

        if(i == maxLevel-1)
            lastLevel = 1;

        reserveMemoryForNodes <<< blockCountReserve, threadPerBlockReserve >>> (nodePool, maxNodes, i, d_counter, volumeResolution, 3, lastLevel);
        printf("memory reserved level %d\n", i);
        hipDeviceSynchronize();

        hipMemcpy(h_counter, d_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);

        printf("reserved node tiles: %d\n", *h_counter);
    }
    //insertVoxelsInLastLevel(node *nodePool, uint1 *positionBuffer, uchar4* colorBufferDevPointer, unsigned int maxLevel)
    hipDeviceSynchronize();
    insertVoxelsInLastLevel<<<blockCount,threadsPerBlock>>>(nodePool,positionDevPointer,colorBufferDevPointer,maxLevel, fragmentListSize);

    hipFree(d_counter);
    delete h_counter;

    return errorCode;
}

hipError_t clearNodePoolCuda(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 64;
    int blockCount = poolSize / threadsPerBlock;

    clearNodePoolKernel<<<blockCount, threadsPerBlock>>>(nodePool, poolSize);
    printf("memory cleared\n");

    return errorCode;
}