#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <src/Utilities/errorUtils.h>
#include "fillOctree.cuh"


const int maxNodePoolSize = 1024;
int volumeResolution = 384;

bool constantMemoryValid = false;   // the flag indicates wheather a kernel is allowed to use the constantNodePool
__constant__ node constNodePool[maxNodePoolSize];
__constant__ int constVolumeResolution[1];
__device__ unsigned int globalNodePoolCounter = 0;

surface<void, cudaSurfaceType3D> surfRef;

hipError_t setVolumeResulution(int resolution)
{
    volumeResolution = resolution;
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constVolumeResolution), &resolution, sizeof(int));
    return errorCode;
}

__device__
unsigned int getBit(unsigned int value, unsigned int position)
{
    return (value >> (position-1)) & 1u;
}

__device__
void setBit(unsigned int &value, unsigned int position)
{
    value |= (1u << (position-1));
}

__device__
void unSetBit(unsigned int &value, unsigned int position)
{
    value &= ~(1u << (position-1));
}

__global__
void testFilling(dim3 texture_dim)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if(x >= texture_dim.x || y >= texture_dim.y || z >= texture_dim.z)
    {
        return;
    }

    uchar4 element = make_uchar4(255, 255, 255, 255);
    surf3Dwrite(element, surfRef, x*sizeof(uchar4), y, z);
}

__global__
void clearNodePoolKernel(node *nodePool, int poolSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= poolSize)
        return;

    nodePool[i].nodeTilePointer = 0;
    nodePool[i].value = 0;
}

__global__
void clearCounter()
{
    globalNodePoolCounter = 0;
}

__global__ void markNodeForSubdivision(node *nodePool, int poolSize, int maxLevel, uint1* positionBuffer, int volumeSideLength)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // mask to get 10 bit position coords
    const unsigned int mask_bits = 0x000003FF;
    unsigned int codedPosition = positionBuffer[index].x;

    float3 position;
    // dont forget the .f for casting reasons :P
    position.x = ((codedPosition) & (mask_bits)) / 1024.f;
    position.y = ((codedPosition >> 10) & (mask_bits)) / 1024.f;
    position.z = ((codedPosition >> 20) & (mask_bits)) / 1024.f;

    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    for(int i=0;i<=maxLevel;i++)
    {
        uint3 nextOctant = make_uint3(0, 0, 0);
        // determine octant for the given voxel
        nextOctant.x = static_cast<unsigned int>(2 * position.x);
        nextOctant.y = static_cast<unsigned int>(2 * position.y);
        nextOctant.z = static_cast<unsigned int>(2 * position.z);

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2*nextOctant.y + 4*nextOctant.z;

        // the maxdivide bit indicates wheather the node has children 1 means has children 0 means does not have children
        unsigned int nodeTile = nodePool[nodeOffset+childPointer*8].nodeTilePointer;
        __syncthreads();
        unsigned int maxDivide = getBit(nodeTile,32);

        if(maxDivide == 0)
        {
            // as the node has no children we set the second bit to 1 which indicates that memory should be allocated
            setBit(nodeTile,31); // possible race condition but it is not importatnt in our case
            nodePool[nodeOffset+childPointer*8].nodeTilePointer = nodeTile;
            __syncthreads();
            break;
        }
        else
        {
            // if the node has children we read the pointer to the next nodetile
            childPointer = nodeTile & 0x3fffffff;
        }

        position.x = 2*position.x - nextOctant.x;
        position.y = 2*position.y - nextOctant.y;
        position.z = 2*position.z - nextOctant.z;
    }
}

__global__ void reserveMemoryForNodes(node *nodePool, int maxNodes, int level, unsigned int* counter)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= maxNodes)
        return;

    /*
    double3 position;
    // make sure we traverse all nodes => the position is between 0 and 1
    unsigned int sideLength = static_cast<unsigned int>(cbrtf(powf(8,level)));


    position.x = (index / sideLength*sideLength)/sideLength;
    position.y = ((index / sideLength) % sideLength)/sideLength;
    position.z = (index % sideLength) / sideLength;
     */

    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    uint3 octants[8];
    octants[0] = make_uint3(0,0,0);
    octants[1] = make_uint3(0,0,1);
    octants[2] = make_uint3(0,1,0);
    octants[3] = make_uint3(0,1,1);
    octants[4] = make_uint3(1,0,0);
    octants[5] = make_uint3(1,0,1);
    octants[6] = make_uint3(1,1,0);
    octants[7] = make_uint3(1,1,1);

    for(int i=0;i<=level;i++)
    {
        uint3 nextOctant = octants[index/static_cast<unsigned int>(pow(8.f, static_cast<float>(i))) % 8];

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2*nextOctant.y + 4*nextOctant.z;

        unsigned int pointer = nodePool[nodeOffset+childPointer*8].nodeTilePointer;
        __syncthreads();    //make sure all threads have a valid nodeTilePointer
        unsigned int reserve = getBit(pointer,31);

        if(reserve == 1)
        {
            // increment the global nodecount and allocate the memory in our
            unsigned int adress = atomicAdd(counter,1)+1;

            pointer = (adress & 0x3fffffff) | pointer;

            // set the divide flag to 1. this indicates that the child pointer is valid
            setBit(pointer,32);

            // make sure we don't reserve the same nodeTile next time :)
            unSetBit(pointer,31);

            nodePool[nodeOffset+childPointer*8].nodeTilePointer = pointer;
            __syncthreads();
            break;
        }
        else
        {
            // traverse further
            childPointer = pointer & 0x3fffffff;
        }
    }
}

hipError_t updateBrickPool(hipArray_t &brickPool, dim3 textureDim)
{
    hipError_t errorCode;

    hipChannelFormatDesc channelDesc;
    errorCode = hipGetChannelDesc(&channelDesc, brickPool);

    if(errorCode != hipSuccess)
        return errorCode;

    errorCode = cudaBindSurfaceToArray(&surfRef, brickPool, &channelDesc);

    if(errorCode != hipSuccess)
        return errorCode;

    dim3 block_dim(4, 4, 4);
    dim3 grid_dim(textureDim.x/block_dim.x, textureDim.y/block_dim.y, textureDim.z/block_dim.z);
    testFilling<<<grid_dim, block_dim>>>(textureDim);

    return hipSuccess;
}

hipError_t updateNodePool(uchar4* colorBufferDevPointer, node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 64;
    int blockCount = poolSize / threadsPerBlock;


    struct node *node_h = (struct node*)malloc(sizeof(node) * poolSize);

    errorCode = hipMemcpy(node_h, nodePool, sizeof(node) * poolSize, hipMemcpyDeviceToHost);

    if(errorCode != hipSuccess)
        return errorCode;


    free(node_h);

    return hipSuccess;
}

hipError_t copyNodePoolToConstantMemory(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constNodePool),nodePool,sizeof(node)*poolSize,0,hipMemcpyDeviceToDevice);

    if(errorCode != hipSuccess)
    {
        constantMemoryValid = false;
        return errorCode;
    }
    else
    {
        constantMemoryValid = true;
        return errorCode;
    }
}

hipError_t buildSVO(node *nodePool,
                     int poolSize,
                     hipArray_t *brickPool,
                     dim3 textureDim,
                     uint1* positionDevPointer,
                     uchar4* colorBufferDevPointer,
                     uchar4* normalDevPointer,
                     int fragmentListSize)
{
    hipError_t errorCode = hipSuccess;
    // calculate maxlevel
    int maxLevel = static_cast<int>(log((volumeResolution*volumeResolution*volumeResolution)/27)/log(8));

    printf("max level: %d \n", maxLevel);

    dim3 block_dim(32, 0, 0);
    dim3 grid_dim(fragmentListSize/block_dim.x, 0, 0);

    int threadsPerBlock = 64;
    int blockCount = fragmentListSize / threadsPerBlock;


    unsigned int *h_counter = new unsigned int[1];
    unsigned int *d_counter;
    *h_counter = 0;

    hipMalloc(&d_counter, sizeof(int));
    hipMemcpy(d_counter,h_counter,sizeof(unsigned int),hipMemcpyHostToDevice);

    clearCounter<<<1,1>>>();
    hipDeviceSynchronize();

    printf("counter: %d\n", *h_counter);

    for(int i=0;i<maxLevel;i++)
    {
        markNodeForSubdivision<<<blockCount, threadsPerBlock>>>(nodePool, poolSize, i, positionDevPointer, 1);
        hipDeviceSynchronize();
        unsigned int maxNodes = static_cast<unsigned int>(pow(8,i));

        const int threadPerBlockReserve = 32;
        int blockCountReserve = maxNodes;

        if(maxNodes >= threadPerBlockReserve)
            blockCountReserve = maxNodes / threadPerBlockReserve;

        reserveMemoryForNodes <<< blockCountReserve, threadPerBlockReserve >>> (nodePool, maxNodes, i, d_counter);
        printf("memory reserved level %d\n", i);
        hipDeviceSynchronize();

        hipMemcpy(h_counter, d_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);

        printf("reserved node tiles: %d\n", *h_counter);
    }

    hipFree(d_counter);
    delete h_counter;

    return errorCode;
}

hipError_t clearNodePoolCuda(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 64;
    int blockCount = poolSize / threadsPerBlock;

    clearNodePoolKernel<<<blockCount, threadsPerBlock>>>(nodePool, poolSize);
    printf("memory cleared\n");

    return errorCode;
}