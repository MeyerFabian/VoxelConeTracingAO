#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <src/Utilities/errorUtils.h>
#include "fillOctree.cuh"


const int maxNodePoolSize = 1024;
int volumeResolution = 384;

bool constantMemoryValid = false;   // the flag indicates wheather a kernel is allowed to use the constantNodePool
__constant__ node constNodePool[maxNodePoolSize];
__constant__ int constVolumeResolution[1];
__device__ unsigned int globalNodePoolCounter = 0;
__device__ unsigned int globalBrickPoolCounter = 0;

surface<void, cudaSurfaceType3D> surfRef;

hipError_t setVolumeResulution(int resolution)
{
    volumeResolution = resolution;
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constVolumeResolution), &resolution, sizeof(int));
    return errorCode;
}

__device__
unsigned int getBit(unsigned int value, unsigned int position)
{
    return (value >> (position-1)) & 1u;
}

__device__
void setBit(unsigned int &value, unsigned int position)
{
    value |= (1u << (position-1));
}

__device__
void unSetBit(unsigned int &value, unsigned int position)
{
    value &= ~(1u << (position-1));
}

__global__
void testFilling(dim3 texture_dim)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if(x >= texture_dim.x || y >= texture_dim.y || z >= texture_dim.z)
    {
        return;
    }

    uchar4 element = make_uchar4(255, 255, 255, 255);
    surf3Dwrite(element, surfRef, x*sizeof(uchar4), y, z);
}

__global__
void clearNodePoolKernel(node *nodePool, int poolSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= poolSize)
        return;

    nodePool[i].nodeTilePointer = 0;
    nodePool[i].value = 0;
}

__global__
void clearCounter()
{
    globalNodePoolCounter = 0;
    globalBrickPoolCounter = 0;
}

__device__ uint3 getBrickCoords(unsigned int brickAdress, unsigned int brickPoolSideLength, unsigned int brickSideLength = 3)
{
    uint3 coords;
    coords.x = brickAdress / (brickPoolSideLength*brickPoolSideLength);
    coords.y = (brickAdress / brickPoolSideLength) % brickPoolSideLength;
    coords.z = brickAdress % brickPoolSideLength;

    coords.x = coords.x*brickSideLength;
    coords.y = coords.y*brickSideLength;
    coords.z = coords.z*brickSideLength;

    return coords;
}

__device__ unsigned int encodeBrickCoords(uint3 coords)
{
    return (0x000003FF & coords.x) << 20U | (0x000003FF & coords.y) << 10U | (0x000003FF & coords.z);
}

__device__ uint3 decodeBrickCoords(unsigned int coded)
{
    uint3 coords;
    coords.z = coded & 0x000003FF;
    coords.y = (coded >> 10) & 0x000003FF;
    coords.x = (coded >> 20) & 0x000003FF;
    return coords;
}

__device__ void fillBrickCorners(const uint3 &brickCoords, const float3 &voxelPosition, const uchar4 &color)
{
    uint3 nextOctant;
    nextOctant.x = static_cast<unsigned int>(2 * voxelPosition.x);
    nextOctant.y = static_cast<unsigned int>(2 * voxelPosition.y);
    nextOctant.z = static_cast<unsigned int>(2 * voxelPosition.z);

    unsigned int offset = nextOctant.x + 2 * nextOctant.y + 4 * nextOctant.z;

    // here we have our possible brick corners // TODO: fill them in const memory maybe?
    uint3 insertPositions[8];
    // front corners
    insertPositions[0] = make_uint3(0,0,0);
    insertPositions[1] = make_uint3(2,0,0);
    insertPositions[2] = make_uint3(2,2,0);
    insertPositions[3] = make_uint3(0,2,0);

    //back corners
    insertPositions[4] = make_uint3(0,0,2);
    insertPositions[5] = make_uint3(2,0,2);
    insertPositions[6] = make_uint3(2,2,2);
    insertPositions[7] = make_uint3(0,2,2);

    /*
    if(brickCoords.x == 0 && brickCoords.y == 609 && brickCoords.z == 840) {
        printf("offset : %d\n", offset);
        printf("color r: %d g: %d b: %d\n", static_cast<unsigned int>(color.x), color.y, color.z);
    }*/

    uint3 pos = insertPositions[offset];

    // write the color value to the corner TODO: use a shared counter to prevent race conditions between double list entries in the fragment list
    surf3Dwrite(color, surfRef, pos.x*sizeof(uchar4), pos.y, pos.z);
}

__global__ void insertVoxelsInLastLevel(node *nodePool, uint1 *positionBuffer, uchar4* colorBufferDevPointer, unsigned int maxLevel, int fragmentListSize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= fragmentListSize)
        return;

    const unsigned int mask_bits = 0x000003FF;
    unsigned int codedPosition = positionBuffer[index].x;

    float3 position;
    // dont forget the .f for casting reasons :P
    position.x = ((codedPosition) & (mask_bits)) / 1024.f;
    position.y = ((codedPosition >> 10) & (mask_bits)) / 1024.f;
    position.z = ((codedPosition >> 20) & (mask_bits)) / 1024.f;


    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;
    unsigned int offset=0;
    unsigned int nodeTile = 0;
    unsigned int value = 0;

    for (int i = 0; i < maxLevel; i++)
    {
        uint3 nextOctant = make_uint3(0, 0, 0);
        // determine octant for the given voxel
        if(i != 0)
        {
            nextOctant.x = static_cast<unsigned int>(2 * position.x);
            nextOctant.y = static_cast<unsigned int>(2 * position.y);
            nextOctant.z = static_cast<unsigned int>(2 * position.z);
        }

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2 * nextOctant.y + 4 * nextOctant.z;
        offset = nodeOffset + childPointer * 8;

        nodeTile = nodePool[offset].nodeTilePointer;

        childPointer = nodeTile & 0x3fffffff;

        if(i != 0)
        {
            position.x = 2 * position.x - nextOctant.x;
            position.y = 2 * position.y - nextOctant.y;
            position.z = 2 * position.z - nextOctant.z;
        }
    }

    // now we fill the corners of our bricks at the last level. This level is represented with 8 values inside a brick
    value = nodePool[offset].value;

    if(getBit(value,32) == 1)
    {
        // we have a valid brick => fill it
        uint3 brickCoords = decodeBrickCoords(value);
        fillBrickCorners(brickCoords,position, colorBufferDevPointer[index]);
    }
}

__global__ void markNodeForSubdivision(node *nodePool, int poolSize, int maxLevel, uint1* positionBuffer, int fragmentListSize)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= fragmentListSize)
        return;

    // mask to get 10 bit position coords
    const unsigned int mask_bits = 0x000003FF;
    unsigned int codedPosition = positionBuffer[index].x;

    float3 position;
    // dont forget the .f for casting reasons :P
    position.x = ((codedPosition) & (mask_bits)) / 1024.f;
    position.y = ((codedPosition >> 10) & (mask_bits)) / 1024.f;
    position.z = ((codedPosition >> 20) & (mask_bits)) / 1024.f;


    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    for(int i=0;i<=maxLevel;i++)
    {
        uint3 nextOctant = make_uint3(0, 0, 0);
        // determine octant for the given voxel
        unsigned int offset = 0;
        if(i != 0)
        {
            nextOctant.x = static_cast<unsigned int>(2 * position.x);
            nextOctant.y = static_cast<unsigned int>(2 * position.y);
            nextOctant.z = static_cast<unsigned int>(2 * position.z);

            // make the octant position 1D for the linear memory
            nodeOffset = nextOctant.x + 2 * nextOctant.y + 4 * nextOctant.z;
            offset = nodeOffset + childPointer * 8;
        }

        // the maxdivide bit indicates wheather the node has children 1 means has children 0 means does not have children
        unsigned int nodeTile = nodePool[offset].nodeTilePointer;
        __syncthreads();
        unsigned int maxDivide = getBit(nodeTile,32);

        if(maxDivide == 0)
        {
            // as the node has no children we set the second bit to 1 which indicates that memory should be allocated
            setBit(nodeTile,31); // possible race condition but it is not importatnt in our case
            nodePool[offset].nodeTilePointer = nodeTile;
            __syncthreads();
            break;
        }
        else
        {
            // if the node has children we read the pointer to the next nodetile
            childPointer = nodeTile & 0x3fffffff;
        }

        if(i!=0)
        {
            position.x = 2 * position.x - nextOctant.x;
            position.y = 2 * position.y - nextOctant.y;
            position.z = 2 * position.z - nextOctant.z;
        }
    }
}

__global__ void reserveMemoryForNodes(node *nodePool, int maxNodes, int level, unsigned int* counter, unsigned int brickPoolResolution, unsigned int brickResolution, int lastLevel)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index >= maxNodes)
        return;

    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    uint3 octants[8];
    octants[0] = make_uint3(0,0,0);
    octants[1] = make_uint3(0,0,1);
    octants[2] = make_uint3(0,1,0);
    octants[3] = make_uint3(0,1,1);
    octants[4] = make_uint3(1,0,0);
    octants[5] = make_uint3(1,0,1);
    octants[6] = make_uint3(1,1,0);
    octants[7] = make_uint3(1,1,1);

    uint3 nextOctant;
    unsigned int octantIdx = 0;

    for (int i = 0; i <=level; i++)
    {
        if(i==0)
            octantIdx = 0;
        else
            octantIdx = (index / static_cast<unsigned int>(pow(8.f, static_cast<float>(i-1)))) % 8;

        nextOctant = octants[octantIdx];

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2 * nextOctant.y + 4 * nextOctant.z;

        unsigned int offset = nodeOffset + childPointer * 8;

        unsigned int pointer = nodePool[offset].nodeTilePointer;
        unsigned int value = nodePool[offset].value;
        __syncthreads();    //make sure all threads have a valid nodeTilePointer

        unsigned int reserve = getBit(pointer, 31);
        unsigned int maxDivided = getBit(pointer, 32);
        if (reserve == 1)
        {
            // increment the global nodecount and allocate the memory in our
            unsigned int adress = atomicAdd(counter, 1) + 1;
            unsigned int brickAdress = atomicAdd(&globalBrickPoolCounter, 1);

            pointer = (adress & 0x3fffffff) | pointer;
            value = encodeBrickCoords(getBrickCoords(brickAdress, brickPoolResolution, brickResolution));

            // set the first bit to 1. this indicates, that we use the texture brick instead of a constant value as color.
            setBit(value, 32);
            setBit(pointer, 32);

            // make sure we don't reserve the same nodeTile next time :)
            unSetBit(pointer, 31);

            if(lastLevel == 1)
                unSetBit(pointer,32);

            nodePool[offset].nodeTilePointer = pointer;
            nodePool[offset].value = value;

            __syncthreads();
            break;
        }
        else
        {
            // traverse further
            childPointer = pointer & 0x3fffffff;
        }
    }

}

hipError_t updateBrickPool(hipArray_t &brickPool, dim3 textureDim)
{
    hipError_t errorCode;

    hipChannelFormatDesc channelDesc;
    errorCode = hipGetChannelDesc(&channelDesc, brickPool);

    if(errorCode != hipSuccess)
        return errorCode;

    errorCode = cudaBindSurfaceToArray(&surfRef, brickPool, &channelDesc);

    if(errorCode != hipSuccess)
        return errorCode;

    dim3 block_dim(4, 4, 4);
    dim3 grid_dim(textureDim.x/block_dim.x, textureDim.y/block_dim.y, textureDim.z/block_dim.z);
    testFilling<<<grid_dim, block_dim>>>(textureDim);

    return hipSuccess;
}

hipError_t updateNodePool(uchar4* colorBufferDevPointer, node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 64;
    int blockCount = poolSize / threadsPerBlock;


    struct node *node_h = (struct node*)malloc(sizeof(node) * poolSize);

    errorCode = hipMemcpy(node_h, nodePool, sizeof(node) * poolSize, hipMemcpyDeviceToHost);

    if(errorCode != hipSuccess)
        return errorCode;


    free(node_h);

    return hipSuccess;
}

hipError_t copyNodePoolToConstantMemory(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constNodePool),nodePool,sizeof(node)*poolSize,0,hipMemcpyDeviceToDevice);

    if(errorCode != hipSuccess)
    {
        constantMemoryValid = false;
        return errorCode;
    }
    else
    {
        constantMemoryValid = true;
        return errorCode;
    }
}

hipError_t buildSVO(node *nodePool,
                     int poolSize,
                     hipArray_t *brickPool,
                     dim3 textureDim,
                     uint1* positionDevPointer,
                     uchar4* colorBufferDevPointer,
                     uchar4* normalDevPointer,
                     int fragmentListSize)
{
    hipError_t errorCode = hipSuccess;
    // calculate maxlevel
    int maxLevel = static_cast<int>(log((volumeResolution*volumeResolution*volumeResolution))/log(8));
    // note that we dont calculate +1 as we store 8 voxels per brick

    dim3 block_dim(32, 0, 0);
    dim3 grid_dim(fragmentListSize/block_dim.x, 0, 0);

    int threadsPerBlock = 64;
    int blockCount = fragmentListSize / threadsPerBlock;


    unsigned int *h_counter = new unsigned int[1];
    unsigned int *d_counter;
    *h_counter = 0;

    hipMalloc(&d_counter, sizeof(int));
    hipMemcpy(d_counter,h_counter,sizeof(unsigned int),hipMemcpyHostToDevice);

    clearCounter<<<1,1>>>();
    hipDeviceSynchronize();

    int lastLevel = 0;

    for(int i=0;i<maxLevel;i++)
    {
        markNodeForSubdivision<<<blockCount, threadsPerBlock>>>(nodePool, poolSize, i, positionDevPointer, fragmentListSize);
        hipDeviceSynchronize();
        unsigned int maxNodes = static_cast<unsigned int>(pow(8,i));

        const unsigned int threadPerBlockReserve = 512;
        const unsigned int blocksPerGridDim = 64000;

        int blockCountReserve = maxNodes;

        if(maxNodes >= threadPerBlockReserve)
            blockCountReserve = maxNodes / threadPerBlockReserve;


        if(i == maxLevel-1)
            lastLevel = 1;

        reserveMemoryForNodes <<< blockCountReserve, threadPerBlockReserve >>> (nodePool, maxNodes, i, d_counter, volumeResolution, 3, lastLevel);
        hipDeviceSynchronize();

        //hipMemcpy(h_counter, d_counter, sizeof(unsigned int), hipMemcpyDeviceToHost);

    }
    //insertVoxelsInLastLevel(node *nodePool, uint1 *positionBuffer, uchar4* colorBufferDevPointer, unsigned int maxLevel)
    hipDeviceSynchronize();
    insertVoxelsInLastLevel<<<blockCount,threadsPerBlock>>>(nodePool,positionDevPointer,colorBufferDevPointer,maxLevel, fragmentListSize);

    hipFree(d_counter);
    delete h_counter;

    return errorCode;
}

hipError_t clearNodePoolCuda(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 64;
    int blockCount = poolSize / threadsPerBlock;

    clearNodePoolKernel<<<blockCount, threadsPerBlock>>>(nodePool, poolSize);

    return errorCode;
}