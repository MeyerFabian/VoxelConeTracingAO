#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "fillOctree.cuh"


const int maxNodePoolSize = 8192;

bool constantMemoryValid = false;   // the flag indicates wheather a kernel is allowed to use the constantNodePool
__constant__ node constNodePool[maxNodePoolSize];

surface<void, cudaSurfaceType3D> surfRef;

__global__
void testFilling(dim3 texture_dim)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if(x >= texture_dim.x || y >= texture_dim.y || z >= texture_dim.z)
    {
        return;
    }

    uchar4 element = make_uchar4(255, 255, 255, 255);
    surf3Dwrite(element, surfRef, x*sizeof(uchar4), y, z);
}

__global__
void testNodeFilling(node *nodePool, int poolSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= poolSize)
    {
        return;
    }

    nodePool[i].nodeTilePointer = 10;
    nodePool[i].value = 10;
}

hipError_t updateBrickPool(hipArray_t &brickPool, dim3 textureDim)
{
    hipError_t errorCode;

    hipChannelFormatDesc channelDesc;
    errorCode = hipGetChannelDesc(&channelDesc, brickPool);

    if(errorCode != hipSuccess)
        return errorCode;

    errorCode = cudaBindSurfaceToArray(&surfRef, brickPool, &channelDesc);

    if(errorCode != hipSuccess)
        return errorCode;

    dim3 block_dim(4, 4, 4);
    dim3 grid_dim(textureDim.x/block_dim.x, textureDim.y/block_dim.y, textureDim.z/block_dim.z);
    testFilling<<<grid_dim, block_dim>>>(textureDim);

    return hipSuccess;
}

hipError_t updateNodePool(hipArray_t &voxel, node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 16;
    int blockCount = poolSize / threadsPerBlock;

    testNodeFilling<<<blockCount, threadsPerBlock>>>(nodePool, poolSize);

    struct node *node_h = (struct node*)malloc(sizeof(struct node)*poolSize);

    errorCode = hipMemcpy(node_h, nodePool, sizeof(node)*poolSize, hipMemcpyDeviceToHost);

    if(errorCode != hipSuccess)
        return errorCode;

    /*
    for(int i=0;i<poolSize;i++)
        printf("%d, %d \n",node_h[i].nodeTilePointer,node_h[i].value);
        */

    free(node_h);

    return hipSuccess;
}

hipError_t copyNodePoolToConstantMemory(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constNodePool),nodePool,poolSize*sizeof(node),0,hipMemcpyDeviceToDevice);

    if(errorCode != hipSuccess)
    {
        constantMemoryValid = false;
        return errorCode;
    }
    else
    {
        constantMemoryValid = true;
        return errorCode;
    }
}