#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "fillOctree.cuh"


const int maxNodePoolSize = 1024;

bool constantMemoryValid = false;   // the flag indicates wheather a kernel is allowed to use the constantNodePool
__constant__ node constNodePool[maxNodePoolSize];
__constant__ int constVolumeResolution[1];
__device__ int globalNodePoolCounter = 0;

surface<void, cudaSurfaceType3D> surfRef;

hipError_t setVolumeResulution(int resolution)
{
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constVolumeResolution), &resolution, sizeof(int));
    return errorCode;
}

__device__
int getBits(unsigned int value, int start, int quantity)
{
    const unsigned int mask_bits = 0xffffffff;

    assert(start <= 31);
    if (start > 31)
        return 0;

    if(quantity > 32-start)
        quantity = 32-start;

    return (value >> start) & (mask_bits >> (32 - quantity));
}

__device__
int getBit(unsigned int value, int position)
{
    return (value >> position-1) & 1;
}

__device__
void setBit(unsigned int &value, int position)
{
    value |= (1u << (position-1));
}

__global__
void testFilling(dim3 texture_dim)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;
    int z = blockIdx.z*blockDim.z + threadIdx.z;

    if(x >= texture_dim.x || y >= texture_dim.y || z >= texture_dim.z)
    {
        return;
    }

    uchar4 element = make_uchar4(255, 255, 255, 255);
    surf3Dwrite(element, surfRef, x*sizeof(uchar4), y, z);
}

__global__
void testNodeFilling(node *nodePool, int poolSize, uchar4* colorBufferDevPointer)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i >= poolSize)
    {
        return;
    }

    if(i == 0)
        printf("%d ,%d, %d, %d \n",colorBufferDevPointer[0].x, colorBufferDevPointer[0].y, colorBufferDevPointer[0].z, colorBufferDevPointer[0].w);

    nodePool[i].nodeTilePointer = 10;
    nodePool[i].value = getBits(nodePool[i].nodeTilePointer,31,1);
}

__global__ void markNodeForSubdivision(node *nodePool, int poolSize, int maxLevel, uint1* positionBuffer, int volumeSideLength)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    uint codedPosition = positionBuffer[index].x;
    float3 position;
    position.x = getBits(codedPosition,2,10)  / constVolumeResolution[0];
    position.y = getBits(codedPosition,12,10) / constVolumeResolution[0];
    position.z = getBits(codedPosition,22,10) / constVolumeResolution[0];

    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    for(int i=0;i<maxLevel;i++)
    {
        uint3 nextOctant = make_uint3(0, 0, 0);
        // determine octant for the given voxel
        nextOctant.x = static_cast<unsigned int>(2 * position.x);
        nextOctant.y = static_cast<unsigned int>(2 * position.y);
        nextOctant.z = static_cast<unsigned int>(2 * position.z);

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2*nextOctant.y + 4*nextOctant.z;

        unsigned int maxDivide = getBit(nodePool[nodeOffset+childPointer].nodeTilePointer,32);
        if(maxDivide == 0)
        {
            // set second bit to 1
            setBit(nodePool[nodeOffset+childPointer].nodeTilePointer,31);
            break;
        }
        else
        {
            // traverse further
            childPointer = getBits(nodePool[nodeOffset + childPointer].nodeTilePointer, 2, 30);
        }

        position.x = 2*position.x - nextOctant.x;
        position.y = 2*position.y - nextOctant.y;
        position.z = 2*position.z - nextOctant.z;
    }


}

__global__ void reserveMemoryForNodes(node *nodePool, int poolSize, int level)
{
    int indexX = blockIdx.x * blockDim.x + threadIdx.x;
    int indexY = blockIdx.y * blockDim.y + threadIdx.y;
    int indexZ = blockIdx.z * blockDim.z + threadIdx.z;

    float3 position;
    // make sure we traverse all nodes
    position.x = indexX  / constVolumeResolution[0];
    position.y = indexY / constVolumeResolution[0];
    position.z = indexZ / constVolumeResolution[0];

    unsigned int nodeOffset = 0;
    unsigned int childPointer = 0;

    for(int i=0;i<level;i++)
    {
        uint3 nextOctant = make_uint3(0, 0, 0);
        // determine octant for the given voxel
        nextOctant.x = static_cast<unsigned int>(2 * position.x);
        nextOctant.y = static_cast<unsigned int>(2 * position.y);
        nextOctant.z = static_cast<unsigned int>(2 * position.z);

        // make the octant position 1D for the linear memory
        nodeOffset = nextOctant.x + 2*nextOctant.y + 4*nextOctant.z;

        unsigned int reserve = getBit(nodePool[nodeOffset+childPointer].nodeTilePointer,31);
        if(reserve == 1)
        {
            // increment the global nodecount and allocate the memory in our
            int adress = atomicAdd(&globalNodePoolCounter,1);

            // TODO: reserve memory
            // TODO: increase counter
            // TODO: set child pointer of node
            setBit(nodePool[nodeOffset+childPointer].nodeTilePointer,32);
            break;
        }
        else
        {
            // traverse further
            childPointer = getBits(nodePool[nodeOffset + childPointer].nodeTilePointer, 2, 30);
        }

        position.x = 2*position.x - nextOctant.x;
        position.y = 2*position.y - nextOctant.y;
        position.z = 2*position.z - nextOctant.z;
    }
}


hipError_t updateBrickPool(hipArray_t &brickPool, dim3 textureDim)
{
    hipError_t errorCode;

    hipChannelFormatDesc channelDesc;
    errorCode = hipGetChannelDesc(&channelDesc, brickPool);

    if(errorCode != hipSuccess)
        return errorCode;

    errorCode = cudaBindSurfaceToArray(&surfRef, brickPool, &channelDesc);

    if(errorCode != hipSuccess)
        return errorCode;

    dim3 block_dim(4, 4, 4);
    dim3 grid_dim(textureDim.x/block_dim.x, textureDim.y/block_dim.y, textureDim.z/block_dim.z);
    testFilling<<<grid_dim, block_dim>>>(textureDim);

    return hipSuccess;
}

hipError_t updateNodePool(uchar4* colorBufferDevPointer, node *nodePool, int poolSize)
{
    hipError_t errorCode = hipSuccess;
    int threadsPerBlock = 64;
    int blockCount = poolSize / threadsPerBlock;

    testNodeFilling<<<blockCount, threadsPerBlock>>>(nodePool, poolSize, colorBufferDevPointer);

    struct node *node_h = (struct node*)malloc(sizeof(node) * poolSize);

    errorCode = hipMemcpy(node_h, nodePool, sizeof(node) * poolSize, hipMemcpyDeviceToHost);

    if(errorCode != hipSuccess)
        return errorCode;

/*
    for(int i=0;i<poolSize;i++)
        printf("%d, %d \n",node_h[i].nodeTilePointer,node_h[i].value);
*/

    free(node_h);

    return hipSuccess;
}

hipError_t copyNodePoolToConstantMemory(node *nodePool, int poolSize)
{
    hipError_t errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constNodePool),nodePool,sizeof(node)*poolSize,0,hipMemcpyDeviceToDevice);

    if(errorCode != hipSuccess)
    {
        constantMemoryValid = false;
        return errorCode;
    }
    else
    {
        constantMemoryValid = true;
        return errorCode;
    }
}

hipError_t buildSVO(node *nodePool,
                     int poolSize,
                     hipArray_t *brickPool,
                     dim3 textureDim,
                     uint1* positionDevPointer,
                     uchar4* colorBufferDevPointer,
                     uchar4* normalDevPointer,
                     int fragmentListSize)
{
    int maxLevel = 2;
    dim3 block_dim(32, 0, 0);
    dim3 grid_dim(fragmentListSize/block_dim.x, 0, 0);

    int threadsPerBlock = 64;
    int blockCount = fragmentListSize / threadsPerBlock;

    for(int i=0;i<6;i++)
    {
        markNodeForSubdivision<<<blockCount, threadsPerBlock>>>(nodePool, poolSize, maxLevel, positionDevPointer, 1);
        hipDeviceSynchronize();
        unsigned int maxNodes = static_cast<unsigned int>(pow(8,i));
        dim3 nodes(maxNodes,maxNodes,maxNodes);
        // reserve memory
        dim3 block_dim_memory(4, 4, 4);
        dim3 grid_dim_memory(1,1,1);
        if(maxNodes >= 8)
            grid_dim_memory = dim3(nodes.x/block_dim_memory.x, nodes.y/block_dim_memory.y, nodes.z/block_dim_memory.z);

        // start for every possible node in this level a thread. this way we make sure, that we dont miss one
        //reserveMemoryForNodes<<<grid_dim_memory, block_dim_memory>>>(nodePool, poolSize, i);
        //hipDeviceSynchronize();
    }

}